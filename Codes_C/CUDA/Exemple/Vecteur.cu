
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define initTimer struct timeval tv1, tv2; struct timezone tz
#define startTimer gettimeofday(&tv1, &tz)
#define stopTimer gettimeofday(&tv2, &tz)
#define tpsCalcul (tv2.tv_sec-tv1.tv_sec)*1000000L + (tv2.tv_usec-tv1.tv_usec)


#define MAX_DIM_GRID 65535
#define MAX_DIM_BLOCK 1024


long tailleVecteur ;

/* KERNEL CUDA */

__global__ void add_vec_scalaire_gpu(int *vec, int *res, int a, long N) {
	long i = (long)blockIdx.x * (long)blockDim.x + (long)threadIdx.x;
	if (i < N) {
		res[i] = vec[i] + a;
	}
}

void add_vec_scalaire_cpu(int *vec, int *res, int a, long N) 
{
	int i ;
	for (i=0 ; i < N ; i ++) {
		res[i] = vec[i] + a;
	}
}

int main(int argc, char *argv[]) {
	int alpha = 10;
	if (argc < 2) {
		printf("Erreur, manque un argument\n");
		exit(0);
	}
	tailleVecteur = atol(argv[1]);
	long blocksize = 1;	
	if (argc ==3) {
		blocksize = atoi(argv[2]);
	}

	int *vecteur;
	int *resultat;
	int *cudaVec;
	int *cudaRes;

	initTimer;

	long size = sizeof(int)*tailleVecteur;

	vecteur = (int *)malloc(size);
	resultat = (int *)malloc(size);

	if (vecteur == NULL) {
		printf("Allocation memoire qui pose probleme (vecteur) \n");
	}
	if (resultat == NULL) {
		printf("Allocation memoire qui pose probleme (resultat) \n");
	}

	long i ;
	for (i= 0 ; i < tailleVecteur ; i++) {
		vecteur[i] = rand() % 100;
		resultat[i] = 0;
	}

/*	cudaSetDevice(1);	*/

	if (hipMalloc((void **)&cudaVec, size) == hipErrorOutOfMemory) {
		printf("Allocation memoire qui pose probleme (cudaVec) \n");
	}
	if (hipMalloc((void **)&cudaRes, size)  == hipErrorOutOfMemory) {
		printf("Allocation memoire qui pose probleme (cudaRes) \n");
	}

	long dimBlock = blocksize;
	long dimGrid = tailleVecteur/blocksize;
	if ((tailleVecteur % blocksize) != 0) {
		dimGrid++;
	}

	




	// Ici on prend le temps de copy des vecteurs du CPU au GPU

startTimer;
	int res = hipMemcpy(&cudaVec[0], &vecteur[0], size, hipMemcpyHostToDevice);
stopTimer; 

	printf("Copy CPU -> GPU %d \n",res);
	int tpsCPUtoGPU = tpsCalcul;
	printf("Temps de copie CPU->GPU : %d \n", tpsCPUtoGPU);
	

	// Ici on prend le temps de calcul 
startTimer;
	add_vec_scalaire_gpu<<<dimGrid, dimBlock>>>(cudaVec, cudaRes, alpha, tailleVecteur);
stopTimer;


	int tpsCalculGPU = tpsCalcul;
	printf("Temps de calcul GPU : %d \n", tpsCalculGPU);
	

	// Ici on prend le temps de copy des vecteurs du GPU au CPU
startTimer;
	hipMemcpy(&resultat[0], &cudaRes[0], size, hipMemcpyDeviceToHost);
stopTimer;

	int tpsGPUtoCPU = tpsCalcul;
	printf("Temps de copie GPU->CPU : %d \n", tpsGPUtoCPU);
	

	




	/* Test bon fonctionnement */

	bool ok = true;
	int indice = -1;
	for (i= 0 ; i < tailleVecteur ; i++) {
/*		printf("Resultat GPU %d     Resultat CPU %d \n",resultat[i], vecteur[i]+alpha);	*/
		if (resultat[i] != vecteur[i] + alpha) {
			ok = false;
			if (indice ==-1) {
				indice = i;
			}
		}
	}
	printf("------ ");
	printf("dimGrid %ld dimBlock %ld ",dimGrid, dimBlock);
	if (ok) {
		printf("Resultat ok\n");
	} else {
		printf("resultat NON ok (%d)\n", indice);
	}

	int tpsTotal = tpsCPUtoGPU + tpsCalculGPU + tpsGPUtoCPU;
	printf("Vecteur %ld => Temps total pour le GPU = %d \n", tailleVecteur, tpsTotal);

	// Temps de calcul pour le CPU
startTimer;
	add_vec_scalaire_cpu (vecteur, resultat, alpha, tailleVecteur);
stopTimer;
	printf("Vecteur %ld => Temps total CPU : %ld \n", tailleVecteur, tpsCalcul);

	hipFree(cudaVec);
	hipFree(cudaRes);
}



